#include "hip/hip_runtime.h"
#include "decode_yolov10.h"

__global__ void decode_yolov10_device_kernel(int batch_size, int num_class,
                                             int topK, float conf_thresh,
                                             float *src, int srcWidth,
                                             int srcHeight, int srcArea,
                                             float *dst, int dstWidth,
                                             int dstHeight, int dstArea) {
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= srcHeight || dy >= batch_size) {
    return;
  }
  float *pitem = src + dy * srcArea + dx * srcWidth;
  float x1 = *pitem++;
  float y1 = *pitem++;
  float x2 = *pitem++;
  float y2 = *pitem++;
  float confidence = *pitem++;
  int label = int(*pitem++);

  if (confidence < conf_thresh) {
    return;
  }

  int index = atomicAdd(dst + dy * dstArea, 1);
  if (index >> topK) {
    return;
  }
  float *pout_item = dst + dy * dstArea + 1 + index * dstWidth;
  *pout_item++ = x1;
  *pout_item++ = y1;
  *pout_item++ = x2;
  *pout_item++ = y2;
  *pout_item++ = confidence;
  *pout_item++ = label;
  //*pout_item++ = 1;
}

void yolov10::decodeDevice(utils::InitParameter param, float *src, int srcWidth,
                           int srcHeight, int srcArea, float *dst, int dstWidth,
                           int dstHeight) {
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size((srcHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
  int dstArea = 1 + dstWidth * dstHeight;

  decode_yolov10_device_kernel<<<grid_size, block_size, 0, nullptr>>>(
      param.batch_size, param.num_class, param.topK, param.conf_thresh, src,
      srcWidth, srcHeight, srcArea, dst, dstWidth, dstHeight, dstArea);
}

__global__ void transpose_device_kernel(int batch_size, float *src,
                                        int srcWidth, int srcHeight,
                                        int srcArea, float *dst, int dstWidth,
                                        int dstHeight, int dstArea) {
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= dstHeight || dy >= batch_size) {
    return;
  }
  float *p_dst_row = dst + dy * dstArea + dx * dstWidth;
  float *p_src_col = src + dy * srcArea + dx;

  for (int i = 0; i < dstWidth; i++) {
    p_dst_row[i] = p_src_col[i * srcWidth];
  }
}

void yolov10::transposeDevice(utils::InitParameter param, float *src,
                              int srcWidth, int srcHeight, int srcArea,
                              float *dst, int dstWidth, int dstHeight) {
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size((dstHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
  int dstArea = dstWidth * dstHeight;

  transpose_device_kernel<<<grid_size, block_size, 0, nullptr>>>(
      param.batch_size, src, srcWidth, srcHeight, srcArea, dst, dstWidth,
      dstHeight, dstArea);
}
