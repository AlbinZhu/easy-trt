#include "hip/hip_runtime.h"
#include "decode_yolov8.h"

__global__ void decode_yolov8_device_kernel(int batch_size, int num_class,
                                            int topK, float conf_thresh,
                                            float *src, int srcWidth,
                                            int srcHeight, int srcArea,
                                            float *dst, int dstWidth,
                                            int dstHeight, int dstArea) {
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= srcHeight || dy >= batch_size) {
    return;
  }
  float *pitem = src + dy * srcArea + dx * srcWidth;
  float *class_confidence = pitem + 4;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_class; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }
  if (confidence < conf_thresh) {
    return;
  }
  int index = atomicAdd(dst + dy * dstArea, 1);

  if (index >= topK) {
    return;
  }
  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;

  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  float *pout_item = dst + dy * dstArea + 1 + index * dstWidth;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;
}

void yolov8::decodeDevice(utils::InitParameter param, float *src, int srcWidth,
                          int srcHeight, int srcArea, float *dst, int dstWidth,
                          int dstHeight) {
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size((srcHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
  int dstArea = 1 + dstWidth * dstHeight;

  decode_yolov8_device_kernel<<<grid_size, block_size, 0, nullptr>>>(
      param.batch_size, param.num_class, param.topK, param.conf_thresh, src,
      srcWidth, srcHeight, srcArea, dst, dstWidth, dstHeight, dstArea);
}

__global__ void transpose_device_kernel(int batch_size, float *src,
                                        int srcWidth, int srcHeight,
                                        int srcArea, float *dst, int dstWidth,
                                        int dstHeight, int dstArea) {
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= dstHeight || dy >= batch_size) {
    return;
  }
  float *p_dst_row = dst + dy * dstArea + dx * dstWidth;
  float *p_src_col = src + dy * srcArea + dx;

  for (int i = 0; i < dstWidth; i++) {
    p_dst_row[i] = p_src_col[i * srcWidth];
  }
}

void yolov8::transposeDevice(utils::InitParameter param, float *src,
                             int srcWidth, int srcHeight, int srcArea,
                             float *dst, int dstWidth, int dstHeight) {
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size((dstHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (param.batch_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
  int dstArea = dstWidth * dstHeight;

  transpose_device_kernel<<<grid_size, block_size, 0, nullptr>>>(
      param.batch_size, src, srcWidth, srcHeight, srcArea, dst, dstWidth,
      dstHeight, dstArea);
}
