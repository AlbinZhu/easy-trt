
#include <hip/hip_runtime.h>
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef WITH_GPU
#include "fastdeploy/vision/common/processors/normalize.h"

namespace fastdeploy {
namespace vision {

__global__ void NormalizeKernel(const uint8_t* src, float* dst,
                                const float* alpha, const float* beta,
                                int num_channel, bool swap_rb, int batch_size,
                                int edge) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= edge) return;

  int img_size = edge / batch_size;
  int n = idx / img_size;        // batch index
  int p = idx - (n * img_size);  // pixel index within the image

  for (int i = 0; i < num_channel; ++i) {
    int j = i;
    if (swap_rb) {
      j = 2 - i;
    }
    dst[num_channel * idx + j] =
        src[num_channel * idx + j] * alpha[i] + beta[i];
  }
}

bool Normalize::ImplByCuda(FDMat* mat) {
  if (mat->layout != Layout::HWC) {
    FDERROR << "The input data must be NHWC format!" << std::endl;
    return false;
  }

  // Prepare input tensor
  FDTensor* src = CreateCachedGpuInputTensor(mat);
  src->ExpandDim(0);
  FDMatBatch mat_batch;
  mat_batch.SetTensor(src);
  mat_batch.mat_type = ProcLib::CUDA;
  mat_batch.input_cache = mat->input_cache;
  mat_batch.output_cache = mat->output_cache;

  bool ret = ImplByCuda(&mat_batch);

  FDTensor* dst = mat_batch.Tensor();
  dst->Squeeze(0);
  mat->SetTensor(dst);
  mat->mat_type = ProcLib::CUDA;
  return true;
}

bool Normalize::ImplByCuda(FDMatBatch* mat_batch) {
  if (mat_batch->layout != FDMatBatchLayout::NHWC) {
    FDERROR << "The input data must be NHWC format!" << std::endl;
    return false;
  }
  // Prepare input tensor
  FDTensor* src = CreateCachedGpuInputTensor(mat_batch);

  // Prepare output tensor
  mat_batch->output_cache->Resize(src->Shape(), FDDataType::FP32,
                                  "batch_output_cache", Device::GPU);

  // Copy alpha and beta to GPU
  gpu_alpha_.Resize({1, 1, static_cast<int>(alpha_.size())}, FDDataType::FP32,
                    "alpha", Device::GPU);
  cudaMemcpy(gpu_alpha_.Data(), alpha_.data(), gpu_alpha_.Nbytes(),
             cudaMemcpyHostToDevice);

  gpu_beta_.Resize({1, 1, static_cast<int>(beta_.size())}, FDDataType::FP32,
                   "beta", Device::GPU);
  cudaMemcpy(gpu_beta_.Data(), beta_.data(), gpu_beta_.Nbytes(),
             cudaMemcpyHostToDevice);

  int jobs =
      mat_batch->output_cache->Numel() / mat_batch->output_cache->shape[3];
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  NormalizeKernel<<<blocks, threads, 0, mat_batch->Stream()>>>(
      reinterpret_cast<uint8_t*>(src->Data()),
      reinterpret_cast<float*>(mat_batch->output_cache->Data()),
      reinterpret_cast<float*>(gpu_alpha_.Data()),
      reinterpret_cast<float*>(gpu_beta_.Data()),
      mat_batch->output_cache->shape[3], swap_rb_,
      mat_batch->output_cache->shape[0], jobs);

  mat_batch->SetTensor(mat_batch->output_cache);
  mat_batch->mat_type = ProcLib::CUDA;
  return true;
}

#ifdef ENABLE_CVCUDA
bool Normalize::ImplByCvCuda(FDMat* mat) { return ImplByCuda(mat); }

bool Normalize::ImplByCvCuda(FDMatBatch* mat_batch) {
  return ImplByCuda(mat_batch);
}
#endif

}  // namespace vision
}  // namespace fastdeploy
#endif
