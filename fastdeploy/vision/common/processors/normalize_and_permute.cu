
#include <hip/hip_runtime.h>
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef WITH_GPU
#include "fastdeploy/vision/common/processors/normalize_and_permute.h"

namespace fastdeploy {
namespace vision {

__global__ void NormalizeAndPermuteKernel(const uint8_t* src, float* dst,
                                          const float* alpha, const float* beta,
                                          int num_channel, bool swap_rb,
                                          int batch_size, int edge) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= edge) return;

  int img_size = edge / batch_size;
  int n = idx / img_size;        // batch index
  int p = idx - (n * img_size);  // pixel index within the image

  for (int i = 0; i < num_channel; ++i) {
    int j = i;
    if (swap_rb) {
      j = 2 - i;
    }
    dst[n * img_size * num_channel + i * img_size + p] =
        src[num_channel * idx + j] * alpha[i] + beta[i];
  }
}

bool NormalizeAndPermute::ImplByCuda(FDMat* mat) {
  if (mat->layout != Layout::HWC) {
    FDERROR << "Only supports input with HWC layout." << std::endl;
    return false;
  }
  // Prepare input tensor
  FDTensor* src = CreateCachedGpuInputTensor(mat);

  // Prepare output tensor
  mat->output_cache->Resize({src->shape[2], src->shape[0], src->shape[1]},
                            FDDataType::FP32, "output_cache", Device::GPU);

  // Copy alpha and beta to GPU
  gpu_alpha_.Resize({1, 1, static_cast<int>(alpha_.size())}, FDDataType::FP32,
                    "alpha", Device::GPU);
  cudaMemcpy(gpu_alpha_.Data(), alpha_.data(), gpu_alpha_.Nbytes(),
             cudaMemcpyHostToDevice);

  gpu_beta_.Resize({1, 1, static_cast<int>(beta_.size())}, FDDataType::FP32,
                   "beta", Device::GPU);
  cudaMemcpy(gpu_beta_.Data(), beta_.data(), gpu_beta_.Nbytes(),
             cudaMemcpyHostToDevice);

  int jobs = 1 * mat->Width() * mat->Height();
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  NormalizeAndPermuteKernel<<<blocks, threads, 0, mat->Stream()>>>(
      reinterpret_cast<uint8_t*>(src->Data()),
      reinterpret_cast<float*>(mat->output_cache->Data()),
      reinterpret_cast<float*>(gpu_alpha_.Data()),
      reinterpret_cast<float*>(gpu_beta_.Data()), mat->Channels(), swap_rb_, 1,
      jobs);

  mat->layout = Layout::CHW;
  mat->SetTensor(mat->output_cache);
  mat->mat_type = ProcLib::CUDA;
  return true;
}

bool NormalizeAndPermute::ImplByCuda(FDMatBatch* mat_batch) {
  // Prepare input tensor
  FDTensor* src = CreateCachedGpuInputTensor(mat_batch);

  // Prepare output tensor
  mat_batch->output_cache->Resize(src->Shape(), FDDataType::FP32,
                                  "batch_output_cache", Device::GPU);
  // NHWC -> NCHW
  std::swap(mat_batch->output_cache->shape[1],
            mat_batch->output_cache->shape[3]);
  std::swap(mat_batch->output_cache->shape[2],
            mat_batch->output_cache->shape[3]);

  // Copy alpha and beta to GPU
  gpu_alpha_.Resize({1, 1, static_cast<int>(alpha_.size())}, FDDataType::FP32,
                    "alpha", Device::GPU);
  cudaMemcpy(gpu_alpha_.Data(), alpha_.data(), gpu_alpha_.Nbytes(),
             cudaMemcpyHostToDevice);

  gpu_beta_.Resize({1, 1, static_cast<int>(beta_.size())}, FDDataType::FP32,
                   "beta", Device::GPU);
  cudaMemcpy(gpu_beta_.Data(), beta_.data(), gpu_beta_.Nbytes(),
             cudaMemcpyHostToDevice);

  int jobs =
      mat_batch->output_cache->Numel() / mat_batch->output_cache->shape[1];
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);
  NormalizeAndPermuteKernel<<<blocks, threads, 0, mat_batch->Stream()>>>(
      reinterpret_cast<uint8_t*>(src->Data()),
      reinterpret_cast<float*>(mat_batch->output_cache->Data()),
      reinterpret_cast<float*>(gpu_alpha_.Data()),
      reinterpret_cast<float*>(gpu_beta_.Data()),
      mat_batch->output_cache->shape[1], swap_rb_,
      mat_batch->output_cache->shape[0], jobs);

  mat_batch->SetTensor(mat_batch->output_cache);
  mat_batch->layout = FDMatBatchLayout::NCHW;
  mat_batch->mat_type = ProcLib::CUDA;
  return true;
}

#ifdef ENABLE_CVCUDA
bool NormalizeAndPermute::ImplByCvCuda(FDMat* mat) { return ImplByCuda(mat); }

bool NormalizeAndPermute::ImplByCvCuda(FDMatBatch* mat_batch) {
  return ImplByCuda(mat_batch);
}
#endif

}  // namespace vision
}  // namespace fastdeploy
#endif
